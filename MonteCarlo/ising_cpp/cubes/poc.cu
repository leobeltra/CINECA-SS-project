#include "hip/hip_runtime.h"
// data structure for efficiently load and store data.
// It contains a 4x4 half tile of unsigned chars to keep the values of booleans.
// Since the kernels update data in chessboard manner the 4x4 tiles are divided in
// black and white elements, this means that the data structure only contains either black
// for white elements. In order to have all elements of the entire 4x4 tile you need two structures.
#include "../utils/uintOfBits.h"
#include <array>
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdio.h>

#define NUM_PROBABILITIES 12 // 2 (s_i combinations) * 6 (sum s_j combinations)

#define FLIP_WITH_PROBABILITY()                                                                    \
    probability = probabilities[spin_to_update * NUM_PROBABILITIES / 2 + neighbors_sum];           \
    accept = hiprand_uniform(&local_state) < probability;                                           \
    if (accept) {                                                                                  \
        even_tile.c_tlb.el_brf = !even_tile.c_tlb.el_brf; /* Flip the spin */                      \
        local_accepted++;                                                                          \
    }

struct CubeEvenElements { // waisting 4 bits per cube
    union {
        u_int8_t cube : 4;
        struct {
            u_int8_t el_tlf : 1; // top-left-front
            u_int8_t el_trb : 1; // top-right-back
            u_int8_t el_blb : 1; // bottom-left-back
            u_int8_t el_brf : 1; // bottom-right-front
        };
    };
};
struct CubeOddElements { // waisting 4 bits per cube
    union {
        u_int8_t cube : 4;
        struct {
            u_int8_t el_tlb : 1; // top-left-back
            u_int8_t el_trf : 1; // top-right-front
            u_int8_t el_blf : 1; // bottom-left-front
            u_int8_t el_brb : 1; // bottom-right-back
        };
    };
};

template <typename CubeElements>
struct Tile {
    union {
        uint_of_bits<8 * sizeof(CubeElements)> tile;
        struct {
            CubeElements c_tlf; // cube top-left-front
            CubeElements c_tlb; // cube top-left-back
            CubeElements c_trf; // cube top-right-front
            CubeElements c_trb; // cube top-right-back

            CubeElements c_blf; // cube bottom-left-front
            CubeElements c_blb; // cube bottom-left-back
            CubeElements c_brf; // cube bottom-right-front
            CubeElements c_brb; // cube bottom-right-back
        };
    };
};

__global__ void efficientKernel(Tile<CubeEvenElements> *matrix, int Ndiv4) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < Ndiv4) {
        Tile tile = matrix[idx];
        u_int32_t c_tlf_el_brb = (u_int32_t)(tile.c_tlf.el_blb) + (u_int32_t)tile.c_tlf.el_brf +
                                 (u_int32_t)tile.c_tlf.el_trb + (u_int32_t)tile.c_tlb.el_brf +
                                 (u_int32_t)tile.c_blf.el_trb + (u_int32_t)tile.c_trf.el_blb;
        volatile double x = 0.11 * c_tlf_el_brb;
    }
}

#include "../gpu.h"
#include <hip/hip_runtime.h>
#if DEBUG
#include <iostream>
#include <stdio.h>
#endif

#ifndef FAKE_CURAND
#include <hiprand/hiprand_kernel.h>
#else
// Fake CURAND implementation for testing without dependencies
// Simple replacement for hiprandState
struct hiprandState {
    unsigned long long seed;
    unsigned long long sequence;
    unsigned long long offset;
    unsigned int state[16]; // Internal state array
};

// Fake hiprand_init implementation
__device__ void hiprand_init(unsigned long seed,
                            unsigned long long sequence,
                            unsigned long long offset,
                            hiprandState *state) {
    state->seed = seed;
    state->sequence = sequence;
    state->offset = offset;

    // Simple initialization of state array
    for (int i = 0; i < 16; i++) {
        state->state[i] = seed + sequence + i;
    }
}

// Fake hiprand_uniform implementation
__device__ float hiprand_uniform(hiprandState *state) {
    // Very simple LCG random number generator
    state->state[0] = 1664525 * state->state[0] + 1013904223;
    // Return a value between 0 and 1
    return (state->state[0] & 0x7FFFFFFF) / float(0x7FFFFFFF);
}

// Double precision version
__device__ double hiprand_uniform_double(hiprandState *state) {
    state->state[0] = 1664525 * state->state[0] + 1013904223;
    return (state->state[0] & 0x7FFFFFFF) / double(0x7FFFFFFF);
}
#endif

#define BLOCK_SIZE 16 // Define block size for CUDA kernels

// CUDA kernel for initializing random states, TODO: use real_t
__global__ void setup_rand_kernel(hiprandState *state, int N, unsigned long seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;

    if (idx >= N || idy >= N)
        return;

    int index = idy * N + idx;
    hiprand_init(seed, index, 0, &state[index]);
}

// CUDA kernel for Ising model simulation (checkerboard pattern, even sites)
template <typename T>
__global__ void ising_kernel_even(Tile<CubeEvenElements> *__restrict__ spins_even,
                                  Tile<CubeOddElements> const *const __restrict__ spins_odd,
                                  int size_x,
                                  int size_y,
                                  int size_z,
                                  const std::array<T, NUM_PROBABILITIES> probabilities,
                                  hiprandState *rand_states, // one generator per Tile
                                  unsigned long long *__restrict__ accepted) {
    const int idx = blockIdx.x * blockDim.x + threadIdx.x;
    const int idy = blockIdx.y * blockDim.y + threadIdx.y;
    const int idz = blockIdx.z * blockDim.z + threadIdx.z;

    // Skip out of bounds
    if (idx >= size_x || idy >= size_y || idz >= size_z)
        return;

    const int tile_idx = idz * (size_x * size_y) + idy * size_x + idx;
    unsigned long long local_accepted = 0;

    // Load random generator and current spins to update
    hiprandState local_state = rand_states[tile_idx];
    Tile<CubeEvenElements> even_tile = spins_even[tile_idx];

    // Calculate the indices of neighboring tiles with periodic boundary conditions in 3D
    const int left = idy * size_x + ((idx - 1 + size_x) % size_x);
    const int right = idy * size_x + ((idx + 1) % size_x);
    const int up = ((idy - 1 + size_y) % size_y) * size_x + idx;
    const int down = ((idy + 1) % size_y) * size_x + idx;
    const int front = idz * (size_x * size_y) + idy * size_x + idx;
    const int back = ((idz - 1 + size_z) % size_z) * (size_x * size_y) + idy * size_x + idx;

    // Update one spin at a time in the even tile

    // Start by updating the spins that can be updated using only the same tile but odd
    Tile<CubeOddElements> odd_tile_central = spins_odd[tile_idx];

    // Process element bottom-left-back in cube top-right-font (c_trf_el_blb)
    u_int32_t spin_to_update = even_tile.c_trf.el_blb;
    u_int32_t neighbors_sum =
        (u_int32_t)odd_tile_central.c_tlf.el_brb + (u_int32_t)odd_tile_central.c_trb.el_blf +
        (u_int32_t)odd_tile_central.c_trf.el_blf + (u_int32_t)odd_tile_central.c_trf.el_brb +
        (u_int32_t)odd_tile_central.c_trf.el_tlb + (u_int32_t)odd_tile_central.c_brf.el_tlb;
    T probability = probabilities[spin_to_update * NUM_PROBABILITIES / 2 + neighbors_sum];
    bool accept = hiprand_uniform(&local_state) < probability;
    if (accept) {
        even_tile.c_trf.el_blb = !even_tile.c_trf.el_blb; // Flip the spin
        local_accepted++;
    }

    // In cube top-left-back process element bottom-right-front (c_tlb_el_brf)
    spin_to_update = even_tile.c_tlb.el_brf;
    neighbors_sum = (u_int32_t)odd_tile_central.c_tlf.el_brb + // front
                    (u_int32_t)odd_tile_central.c_tlb.el_brb + // back
                    (u_int32_t)odd_tile_central.c_blb.el_trf + // down
                    (u_int32_t)odd_tile_central.c_tlb.el_trf + // up
                    (u_int32_t)odd_tile_central.c_trb.el_blf + // right
                    (u_int32_t)odd_tile_central.c_tlb.el_blf;  // left
    FLIP_WITH_PROBABILITY();

    // In cube bottom-left-front process element top-right-back (c_blf_el_trb)
    spin_to_update = even_tile.c_blf.el_trb;
    neighbors_sum = (u_int32_t)odd_tile_central.c_blf.el_trf + // front
                    (u_int32_t)odd_tile_central.c_blb.el_trf + // back
                    (u_int32_t)odd_tile_central.c_blf.el_brb + // down
                    (u_int32_t)odd_tile_central.c_tlf.el_brb + // up
                    (u_int32_t)odd_tile_central.c_brf.el_tlb + // right
                    (u_int32_t)odd_tile_central.c_blf.el_tlb;  // left
    FLIP_WITH_PROBABILITY();

    // Now update the spins that can be updated using central and above tiles

    // In cube top-right-front process element top-left-front (c_trf_el_tlf)
    spin_to_update = even_tile.c_trf.el_tlf;
    neighbors_sum = (u_int32_t)odd_tile_central.c_trf.el_b + 0; // TODO continue here

    // In cube top-left-back process element bottom-left-front (c_tlb_el_blf)
    spin_to_update = even_tile.c_tlb.el_blf;
    neighbors_sum = (u_int32_t)odd_tile_central.c_tlf.el_b + 0; // TODO continue here

    // In cube bottom-right-back process element top-left-front (c_brb_el_tlf)

    // Update random state
    rand_states[index] = local_state;
}

// CUDA kernel for Ising model simulation (checkerboard pattern, odd sites)
template <typename T>
__global__ void ising_kernel_odd(T *even_spins,
                                 T *odd_spins,
                                 int N,
                                 T beta,
                                 T J,
                                 T B_field,
                                 hiprandState *rand_states,
                                 unsigned long long *accepted) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;

    // Skip even sites in checkerboard pattern
    if ((idx + idy) % 2 == 0)
        return;

    // Skip out of bounds
    if (idx >= N || idy >= N)
        return;

    int index = idy * N + idx;
    hiprandState local_state = rand_states[index];

    // Calculate the indices of neighboring spins with periodic boundary conditions
    int left = idy * N + ((idx - 1 + N) % N);
    int right = idy * N + ((idx + 1) % N);
    int up = ((idy - 1 + N) % N) * N + idx;
    int down = ((idy + 1) % N) * N + idx;

    // Calculate energy change if we flip this spin
    const T current_spin = spins[index];
    T neighbor_sum = spins[left] + spins[right] + spins[up] + spins[down];
    T dE = T{2} * J * current_spin * neighbor_sum + T{2} * B_field * current_spin;

    // Apply Metropolis algorithm
    T acceptance_prob = exp(-beta * dE);
    const bool accept = hiprand_uniform(&local_state) < acceptance_prob;

    if (accept) {
        new_spins[index] = -current_spin;
        spins[index] = -current_spin; // Update the spin state
        atomicAdd(accepted, 1ULL);
    } else {
        new_spins[index] = current_spin;
    }

    // Update random state
    rand_states[index] = local_state;
}

// CUDA kernel for computing partial sums for Ising Hamiltonian
template <typename T>
__global__ void compute_hamiltonian_kernel(const T *spins, T *partial_sums, int N, T J, T B_field) {
    extern __shared__ T shared_data[];

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    int tid = threadIdx.y * blockDim.x + threadIdx.x;

    T h_b = 0.0;
    T h_s_rows = 0.0;
    T h_s_cols = 0.0;

    if (idx < N && idy < N) {
        int index = idy * N + idx;

        // Calculate magnetic field contribution
        h_b = spins[index];

        // Calculate horizontal spin interactions
        h_s_rows = spins[index] * spins[idy * N + ((idx + 1) % N)];

        // Calculate vertical spin interactions
        h_s_cols = spins[index] * spins[((idy + 1) % N) * N + idx];
    }

    // Store partial results in shared memory
    shared_data[tid * 3 + 0] = h_b;
    shared_data[tid * 3 + 1] = h_s_rows;
    shared_data[tid * 3 + 2] = h_s_cols;
    __syncthreads();

    // Parallel reduction within block
    for (int s = blockDim.x * blockDim.y / 2; s > 0; s >>= 1) {
        if (tid < s) {
            shared_data[tid * 3 + 0] += shared_data[(tid + s) * 3 + 0];
            shared_data[tid * 3 + 1] += shared_data[(tid + s) * 3 + 1];
            shared_data[tid * 3 + 2] += shared_data[(tid + s) * 3 + 2];
        }
        __syncthreads();
    }

    // Write block results to global memory
    if (tid == 0) {
        int blockId = blockIdx.y * gridDim.x + blockIdx.x;
        partial_sums[blockId * 3 + 0] = shared_data[0]; // H_B contribution
        partial_sums[blockId * 3 + 1] = shared_data[1]; // H_S_rows contribution
        partial_sums[blockId * 3 + 2] = shared_data[2]; // H_S_cols contribution
    }
}

void simulate_ising_gpu(real_t *spins,
                        real_t *result,
                        int N,
                        int equil_steps,
                        int M_sweep,
                        real_t beta,
                        real_t J,
                        real_t B_field,
                        real_t *acceptancy) {
    // Device memory allocation
    real_t *d_spins, *d_new_spins;
    size_t size = N * N * sizeof(real_t);
    hipMalloc(&d_spins, size);
    hipMalloc(&d_new_spins, size);

    // Copy input spins to device
    hipMemcpy(d_spins, spins, size, hipMemcpyHostToDevice);
    hipMemcpy(d_new_spins, spins, size, hipMemcpyHostToDevice);

    // Allocate memory for random states
    hiprandState *d_rand_states;
    hipMalloc(&d_rand_states, N * N * sizeof(hiprandState));

    // Allocate memory for acceptancy counter
    unsigned long long *d_accepted, h_accepted = 0;
    hipMalloc(&d_accepted, sizeof(unsigned long long));
    real_t temp_acceptancy = 0.0;

    // Setup grid and blocks for CUDA kernels
    dim3 block_size(BLOCK_SIZE, BLOCK_SIZE, 1);
    dim3 grid_size((N + block_size.x - 1) / block_size.x, (N + block_size.y - 1) / block_size.y, 1);

    int num_blocks = grid_size.x * grid_size.y;
    size_t shared_mem_size = block_size.x * block_size.y * 3 * sizeof(real_t);

    // Allocate memory for partial sums
    real_t *d_partial_sums;
    hipMalloc(&d_partial_sums, num_blocks * 3 * sizeof(real_t));
    real_t *h_partial_sums;
    hipHostMalloc(&h_partial_sums, num_blocks * 3 * sizeof(real_t));

    // Initialize random generators
    setup_rand_kernel<<<grid_size, block_size>>>(d_rand_states, N, time(NULL));

#if DEBUG
    // print condition
    auto printCondition = [](int iter) { return (iter % 1 == 0 && iter < 5); };
#endif

    // Equilibration steps
    for (int iter = 0; iter < equil_steps; iter++) {
        hipMemset(d_accepted, 0, sizeof(unsigned long long));
        ising_kernel_even<real_t><<<grid_size, block_size>>>(d_spins, d_new_spins, N, beta, J,
                                                             B_field, d_rand_states, d_accepted);
#if DEBUG
        hipDeviceSynchronize();
        // Check cuda errors
        hipError_t err = hipGetLastError();
        if (err != hipSuccess) {
            std::cerr << "CUDA error in ising_kernel_even: " << hipGetErrorString(err)
                      << std::endl;
            exit(EXIT_FAILURE);
        }
#endif

        ising_kernel_odd<real_t><<<grid_size, block_size>>>(d_new_spins, d_spins, N, beta, J,
                                                            B_field, d_rand_states, d_accepted);

#if DEBUG
        // Copy d_spins to host to print the first 4x4 tile
        if (printCondition(iter)) { // Print every 10 iterations and first 500
            real_t *h_spins = new real_t[N * N];
            hipMemcpy(h_spins, d_spins, size, hipMemcpyDeviceToHost);
            std::cout << "Equilibration step " << iter << ": Spins (first 4x4 tile):" << std::endl;
            for (int i = 0; i < 4; i++) {
                std::cout << "\t";
                for (int j = 0; j < 4; j++) {
                    std::cout << h_spins[i * N + j] << " ";
                }
                std::cout << std::endl;
            }
            delete[] h_spins;
        }
#endif

        // Compute H_issing_2D
        // Launch kernel to compute partial sums
        compute_hamiltonian_kernel<real_t>
            <<<grid_size, block_size, shared_mem_size>>>(d_spins, d_partial_sums, N, J, B_field);

        // Copy partial sums back to host, automatic synchronization
        hipMemcpy(h_partial_sums, d_partial_sums, num_blocks * 3 * sizeof(real_t),
                   hipMemcpyDeviceToHost);

        // Sum up on host (could be optimized with another kernel for large grids)
        real_t h_b_sum = 0.0, h_s_rows_sum = 0.0, h_s_cols_sum = 0.0;
        for (int i = 0; i < num_blocks; i++) {
            h_b_sum += h_partial_sums[i * 3 + 0];
            h_s_rows_sum += h_partial_sums[i * 3 + 1];
            h_s_cols_sum += h_partial_sums[i * 3 + 2];
        }

        // Calculate final Hamiltonian
        real_t h_ising = -B_field * h_b_sum - J * (h_s_rows_sum + h_s_cols_sum);

#if DEBUG
        if (printCondition(iter)) {
            // Print Hamiltonian
            std::cout << "Equilibration step " << iter << ": H_ising = " << h_ising << std::endl;
            // Print acceptance count
            hipMemcpy(&h_accepted, d_accepted, sizeof(unsigned long long), hipMemcpyDeviceToHost);
            hipDeviceSynchronize();
            std::cout << "Acceptance count: " << h_accepted << " N*N/2=" << N * N / 2 << std::endl;
        }
#endif
    }

    // Copy acceptance count to host
    hipMemcpy(&h_accepted, d_accepted, sizeof(unsigned long long), hipMemcpyDeviceToHost);
    temp_acceptancy = (real_t)h_accepted / (real_t)(N * N);

#if DEBUG
    // Print acceptance rate after equilibration
    std::cout << "Acceptance rate after equilibration: " << temp_acceptancy << std::endl;
#endif

    // Measurement steps
    for (int iter = 0; iter < M_sweep; iter++) {

        // Reset acceptance counter for measurement phase
        hipMemset(d_accepted, 0, sizeof(unsigned long long));

        ising_kernel_even<real_t><<<grid_size, block_size>>>(d_spins, d_new_spins, N, beta, J,
                                                             B_field, d_rand_states, d_accepted);
        ising_kernel_odd<real_t><<<grid_size, block_size>>>(d_new_spins, d_spins, N, beta, J,
                                                            B_field, d_rand_states, d_accepted);

        // Copy acceptance count to host, implicit synchronization
        hipMemcpy(&h_accepted, d_accepted, sizeof(unsigned long long), hipMemcpyDeviceToHost);
        temp_acceptancy = (real_t)h_accepted / (real_t)(N * N);
        *acceptancy = *acceptancy + temp_acceptancy;
    }

    // Copy results back to host
    hipMemcpy(result, d_spins, size, hipMemcpyDeviceToHost);
    hipMemcpy(&h_accepted, d_accepted, sizeof(unsigned long long), hipMemcpyDeviceToHost);

    // Calculate acceptancy rate
    *acceptancy = *acceptancy / (real_t)(M_sweep);

    // Free device memory
    hipFree(d_spins);
    hipFree(d_new_spins);
    hipFree(d_rand_states);
    hipFree(d_accepted);
    hipFree(d_partial_sums);
    // Free host memory
    hipHostFree(h_partial_sums);
}
